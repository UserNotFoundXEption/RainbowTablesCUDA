#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstring>
#include <stdint.h> 
#include <getopt.h>
#include "des.cuh"
#include "des_tables.cuh"

#define PW_LEN 8
#define ALPHABET "abcdefghijklmnopqrstuvwxyz"

__device__ __constant__ uint64_t subkeys[16];

int PC1_host[56];
int SHIFTS_host[16];
int PC2_host[48];

__device__ uint64_t reduce(uint64_t block, int round) {
    uint64_t result = 0;
    for (int i = 0; i < PW_LEN; i++) {
        uint8_t byte = (block >> (8 * (i % 8))) & 0xFF;
        int idx = (byte + round + i) % 26;
        result |= ((uint64_t)('a' + idx)) << ((PW_LEN - 1 - i) * 8);
    }
    return result;
}

__global__ void kernel(uint64_t* out, int total_chains, int chain_len) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= total_chains) return;

    uint64_t pw = 0;
    for (int i = 0; i < PW_LEN; i++) {
        pw |= ((uint64_t)'a') << ((PW_LEN - 1 - i) * 8);
    }
    int n = id;
    for (int i = PW_LEN - 1; i >= 0 && n > 0; i--) {
        int shift = (PW_LEN - 1 - i) * 8;
        uint8_t c = ((pw >> shift) & 0xFF) + (n % 26);
        pw = (pw & ~(0xFFULL << shift)) | ((uint64_t)c << shift);
        n /= 26;
    }

    uint64_t start = pw;
    for (int i = 0; i < chain_len; i++) {
        pw = des_encrypt(pw, subkeys);
        pw = reduce(pw, i);
    }

    out[id * 2] = start;
    out[id * 2 + 1] = pw;
}

__global__ void kernel_sequential(uint64_t* out, int total_chains, int chain_len) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        for (int id = 0; id < total_chains; ++id) {
            uint64_t pw = 0;
            for (int i = 0; i < PW_LEN; i++) {
                pw |= ((uint64_t)'a') << ((PW_LEN - 1 - i) * 8);
            }
            int n = id;
            for (int i = PW_LEN - 1; i >= 0 && n > 0; i--) {
                int shift = (PW_LEN - 1 - i) * 8;
                uint8_t c = ((pw >> shift) & 0xFF) + (n % 26);
                pw = (pw & ~(0xFFULL << shift)) | ((uint64_t)c << shift);
                n /= 26;
            }

            uint64_t start = pw;
            for (int i = 0; i < chain_len; i++) {
                pw = des_encrypt(pw, subkeys);
                pw = reduce(pw, i);
            }

            out[id * 2] = start;
            out[id * 2 + 1] = pw;
        }
    }
}

void generate_subkeys(uint64_t key, uint64_t* subkeys) {
    uint64_t perm_key = 0;
    for (int i = 0; i < 56; i++)
        perm_key |= ((key >> (64 - PC1_host[i])) & 1ULL) << (55 - i);

    uint32_t C = (perm_key >> 28) & 0x0FFFFFFF;
    uint32_t D = perm_key & 0x0FFFFFFF;

    for (int i = 0; i < 16; i++) {
        C = ((C << SHIFTS_host[i]) | (C >> (28 - SHIFTS_host[i]))) & 0x0FFFFFFF;
        D = ((D << SHIFTS_host[i]) | (D >> (28 - SHIFTS_host[i]))) & 0x0FFFFFFF;
        uint64_t CD = (((uint64_t)C) << 28) | D;
        uint64_t subkey = 0;
        for (int j = 0; j < 48; j++)
            subkey |= ((CD >> (56 - PC2_host[j])) & 1ULL) << (47 - j);
        subkeys[i] = subkey;
    }
}


int main(int argc, char** argv) {
    bool sequential = false;

    int opt;
    while ((opt = getopt(argc, argv, "s")) != -1) {
        if (opt == 's') sequential = true;
    }

    if (argc - optind < 4) {
        printf("Użycie: %s [-s] <liczba_łańcuchów> <dlugosc_lancucha> <klucz_hex> <watki_na_blok>\n", argv[0]);
        return 1;
    }

    int total_chains = atoi(argv[optind]);
    int chain_len = atoi(argv[optind + 1]);
    uint64_t key = strtoull(argv[optind + 2], NULL, 16);
    int threads_per_block = sequential ? 1 : atoi(argv[optind + 3]);

    hipMemcpyToSymbol(HIP_SYMBOL(PC1), PC1_host, sizeof(PC1_host));
    hipMemcpyToSymbol(HIP_SYMBOL(SHIFTS), SHIFTS_host, sizeof(SHIFTS_host));
    hipMemcpyToSymbol(HIP_SYMBOL(PC2), PC2_host, sizeof(PC2_host));

    uint64_t h_subkeys[16];
    generate_subkeys(key, h_subkeys);
    hipMemcpyToSymbol(HIP_SYMBOL(subkeys), h_subkeys, sizeof(uint64_t) * 16);

    int blocks = sequential ? 1 : (total_chains + threads_per_block - 1) / threads_per_block;

    size_t size = total_chains * 2 * PW_LEN;
    uint64_t* d_out;
    hipMalloc(&d_out, sizeof(uint64_t) * total_chains * 2);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    if (sequential) {
        kernel_sequential<<<1, 1>>>(d_out, total_chains, chain_len);
    } else {
        kernel<<<blocks, threads_per_block>>>(d_out, total_chains, chain_len);
    }

    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA ERROR: %s\n", hipGetErrorString(err));
    }

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    printf("%s działał przez %.4f sekund.\n", sequential ? "Tryb sekwencyjny (GPU, 1 wątek)" : "GPU", ms / 1000.0);

    char* h_out = new char[size];
    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);
    FILE* f = fopen("output/rainbow_des.txt", "w");
    for (int i = 0; i < total_chains; i++) {
        for (int j = 0; j < PW_LEN; j++) fputc(h_out[i * 2 * PW_LEN + j], f);
        fputc(':', f);
        for (int j = 0; j < PW_LEN; j++) fputc(h_out[i * 2 * PW_LEN + PW_LEN + j], f);
        fputc('\n', f);
    }
    fclose(f);
    delete[] h_out;
    hipFree(d_out);
    return 0;
}

